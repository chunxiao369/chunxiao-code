#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "kernel.h"

#define THREADS_NUM 8
// declare the kernel function
__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el);

// function which invokes the kernel
void asum(const float* A, const float* B, float* C, int n_el) {

  // declare the number of blocks per grid and the number of threads per block
  int threadsPerBlock,blocksPerGrid;

  // use 1 to THREADS_NUM threads per block
  printf("n_el: %d.\n", n_el);
  if (n_el< THREADS_NUM){
    threadsPerBlock = n_el;
    blocksPerGrid   = 1;
  } else {
    threadsPerBlock = THREADS_NUM;
    blocksPerGrid   = ceil(double(n_el)/double(threadsPerBlock));
  }
  printf("blocks: %d, threads: %d\n", blocksPerGrid,threadsPerBlock);
  // invoke the kernel
  kernel_sum<<<blocksPerGrid,threadsPerBlock>>>(A, B, C, n_el);
}

// kernel
__global__ void kernel_sum(const float* A, const float* B, float* C, int n_el)
{
  // calculate the unique thread index
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  printf("dim: %d, block: %d, thread: %d.\n", blockDim.x, blockIdx.x, threadIdx.x);
  // perform tid-th elements addition 
  if (tid < n_el) C[tid] = A[tid] + B[tid];
}
