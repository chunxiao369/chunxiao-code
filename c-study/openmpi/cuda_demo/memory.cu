#include <cstdio>
#include <hip/hip_runtime.h>

#define CUDACHECK(cmd)                                              \
  do {                                                              \
    hipError_t err = cmd;                                          \
    if (err != hipSuccess) {                                       \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                              \
      exit(EXIT_FAILURE);                                           \
    }                                                               \
  } while (0)

__global__ void kernel(int *pret) {
    *pret = 42;
}

int main() {
    int *pret;
    CUDACHECK(hipMalloc(&pret, sizeof(int)));
    kernel<<<1, 1>>>(pret);

    int ret;
    CUDACHECK(hipMemcpy(&ret, pret, sizeof(int), hipMemcpyDeviceToHost));
    printf("result: %d\n", ret);

    hipFree(pret);
    return 0;
}