#include <cstdio>
#include <hip/hip_runtime.h>

#define CUDACHECK(cmd)                                              \
  do {                                                              \
    hipError_t err = cmd;                                          \
    if (err != hipSuccess) {                                       \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                              \
      exit(EXIT_FAILURE);                                           \
    }                                                               \
  } while (0)

__global__ void kernel(int *pret) {
    *pret = 666;
}

__global__ void just_print(int *gp) {
    printf("from GPU print vlaue: %d\n", *gp);
}


int main() {
    int *pret;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    CUDACHECK(hipMalloc(&pret, sizeof(int)));
    kernel<<<1, 1>>>(pret);

    int ret;
    CUDACHECK(hipMemcpy(&ret, pret, sizeof(int), hipMemcpyDeviceToHost));
    printf("from CPU print value: %d\n", ret);

    hipEventRecord(start);
    /* GPU calc process*/
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    ret += 222;
    CUDACHECK(hipMemcpy(pret, &ret, sizeof(int), hipMemcpyHostToDevice));
    just_print<<<1, 1>>>(pret);
    //printf("result: %d\n", *pret);
    hipFree(pret);
    return 0;
}
