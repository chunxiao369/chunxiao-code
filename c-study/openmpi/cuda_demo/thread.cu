#include <cstdio>
#include <hip/hip_runtime.h>

__global__ void kernel() {
    /*
    printf("same content: gripDim (%d,%d,%d), blockDIM (%d,%d,%d)\n",
           gridDim.x, gridDim.y, gridDim.z,
           blockDim.x, blockDim.y, blockDim.z);
    */
    printf("Block (%d,%d,%d) , Thread (%d,%d,%d) \n",
           blockIdx.x, blockIdx.y, blockIdx.z,
           threadIdx.x, threadIdx.y, threadIdx.z);
}

int main() {

    /* first para is block number, 2 blocks
     * dim(2,1,1) is three-dimensional number from (0,0,0) -> 1,0,0
     */
    /* second para is thread number per block, 16 threads
     * dim(2,2,4) is three-dimensional number from (0,0,0) -> 1,1,3
     */
    kernel<<<dim3(2, 1, 1), dim3(2, 2, 4)>>>();
    hipDeviceSynchronize();
    return 0;
}

