#include <cstdio>
#include <hip/hip_runtime.h>
/*
#include <helper_cuda.h>
int CUDACores = _ConvertSMVer2Cores(prop.major, prop.minor) * prop.multiProcessorCount;
*/
__device__ void say_hello() {
    printf("Hello, world from GPU!\n");
}

__host__ void say_hello_host() {
    printf("Hello, world from CPU!\n");
}

__global__ void kernel() {
    say_hello();
}

int main() {
    int device_count;
    /* first para is block number*/
    /* second para is thread number per block*/
    kernel<<<1, 2>>>();
    hipDeviceSynchronize();
    say_hello_host();
    hipGetDeviceCount(&device_count);
    for (int i = 0; i < device_count; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device %d: %s, has ", i, prop.name);
        printf("SM Count: %d\n", prop.multiProcessorCount);
    }

    return 0;
}

