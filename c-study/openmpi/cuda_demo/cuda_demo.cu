#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "kernel.h"


// 检查 CUDA API 调用是否成功的宏
#define CUDA_CHECK(call)                                      \
    do {                                                      \
        hipError_t err = call;                               \
        if (err != hipSuccess) {                             \
            fprintf(stderr, "CUDA error at %s:%d code=%d (%s)\n", \
                    __FILE__, __LINE__, err, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

// -------------------------------------------------------------
// CUDA Kernel: 使用共享内存的并行求和
// -------------------------------------------------------------
__global__ void reduce_sum_shared(const int* g_input, int* g_output, int N) {
    // 1. 软件层级：Thread/Block 抽象
    // ------------------------------------------------
    // blockDim.x：线程块中的线程数 (Threads per Block)
    // blockIdx.x：当前线程块的索引 (Block Index)
    // threadIdx.x：当前线程在块内的索引 (Thread Index)
    
    // 线程在整个 Grid 中的全局索引
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (0 == threadIdx.x && 0 == blockIdx.x) {
        printf("grid x blocks: %d, dim blocks: %d, block: %d, thread: %d.\n", gridDim.x, blockDim.x, blockIdx.x, threadIdx.x);
    }
    
    // 线程在块内的局部索引
    int local_tid = threadIdx.x;
    
    // 2. 硬件层级：Shared Memory 和 Bank Conflict 优化
    // ------------------------------------------------
    // __shared__ 声明：数据分配在 SM 上的共享内存中 (非常快速，但容量小)
    // 这种访问模式可以很好地避免 Bank Conflict。
    extern __shared__ int sdata[]; // 动态分配共享内存 (大小在启动时指定)
    //share memory访问延迟约为全局内存（Global Memory）的 1/100，带宽是全局内存的 7~10 倍。

    // 将全局内存数据加载到共享内存
    if (tid < N) {
        sdata[local_tid] = g_input[tid];
    } else {
        // 如果数据量不是 blockDim.x 的整数倍，填充0
        sdata[local_tid] = 0; 
    }
    
    // 同步：确保所有线程都已完成从全局内存到共享内存的加载
    __syncthreads(); 

    // ------------------------------------------------
    // 3. 硬件层级：Warp/SM 的并行归约计算
    // ------------------------------------------------
    // 归约求和：通过并行迭代，将一个 Block 的所有值求和到一个值
    
    // 循环步长 (stride) 从 blockDim.x / 2 开始
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        
        // __syncthreads() 确保每一轮归约完成后，线程才能继续下一轮
        __syncthreads(); 
        
        // 只有线程索引小于步长的线程参与计算
        if (local_tid < stride) {
            // 将 sdata[local_tid + stride] 的值加到 sdata[local_tid]
            // 这就是并行归约的核心操作
            sdata[local_tid] += sdata[local_tid + stride];
            if (0 == threadIdx.x && 0 == blockIdx.x) {
                printf("sum: %d.\n", sdata[0]);
            }
            
            // 硬件配合理解：
            // 第一轮，sdata的后128个各加到前128个。第二轮，sdata的后64个各加到前64个。第三轮后32各加到前32。
            // 在这一步，归约操作由 SM 上的多个 Warp 并行执行。
            // 由于数据在共享内存中，访问速度快，且 sdata[local_tid] 和 sdata[local_tid + stride]
            // 在内存中往往是分散在不同的 Bank，可以实现无 Bank Conflict 的并行访问。
        }
    }
    // 4. 结果写回：将 Block 结果写回全局内存
    // ------------------------------------------------
    // 只有 Block 的第一个线程 (local_tid == 0) 将最终结果写回全局输出数组
    if (local_tid == 0) {
        // g_output 的大小等于 Grid 的大小（即 Block 的数量）
        g_output[blockIdx.x] = sdata[0];
    }
}

// -------------------------------------------------------------
// Host 代码 (CPU)
// -------------------------------------------------------------
int main() {
    // 数组总大小
    //const int N = 1024 * 1024; 
    const int N = 1024 * 8; 
    // 每个线程块的线程数 (通常是 2 的幂，推荐 256/512/1024)
    const int THREADS_PER_BLOCK = 256; 
    // 计算需要的线程块数
    const int NUM_BLOCKS = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK; 
    
    // 申请主机内存
    int* h_input = (int*)malloc(N * sizeof(int));
    
    // 初始化输入数据
    for (int i = 0; i < N; ++i) {
        h_input[i] = 1; // 方便验证，最终结果应为 N
    }

    // 申请设备内存
    int *d_input, *d_output;
    CUDA_CHECK(hipMalloc((void**)&d_input, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_output, NUM_BLOCKS * sizeof(int)));

    // 将输入数据从主机传输到设备
    CUDA_CHECK(hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice));

    // 启动 Kernel
    // 共享内存大小：每个 Block 需要 THREADS_PER_BLOCK * sizeof(int) 的空间
    size_t shared_mem_size = THREADS_PER_BLOCK * sizeof(int); 
    
    // 启动 Kernel，注意第三个参数是动态分配的共享内存大小
    printf("blocks: %d, threads: %d, sharemem: %lu.\n", NUM_BLOCKS, THREADS_PER_BLOCK, shared_mem_size);
    reduce_sum_shared<<<NUM_BLOCKS, THREADS_PER_BLOCK, shared_mem_size>>>(
        d_input, d_output, N
    );
    CUDA_CHECK(hipGetLastError()); // 检查核函数启动是否成功

    // 等待设备完成计算
    CUDA_CHECK(hipDeviceSynchronize());

    // 将部分结果（每个 Block 的求和结果）从设备传输回主机
    int* h_output = (int*)malloc(NUM_BLOCKS * sizeof(int));
    CUDA_CHECK(hipMemcpy(h_output, d_output, NUM_BLOCKS * sizeof(int), hipMemcpyDeviceToHost));

    // 在主机上对 Block 结果进行最终求和
    long long final_sum = 0;
    for (int i = 0; i < NUM_BLOCKS; ++i) {
        final_sum += h_output[i];
    }

    printf("Input size N = %d\n", N);
    printf("Expected Sum = %d\n", N);
    printf("Calculated Sum = %lld\n", final_sum);

    // 清理资源
    free(h_input);
    free(h_output);
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));

    return 0;
}
