#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include "kernel.h"


// 检查 CUDA API 调用是否成功的宏
#define CUDA_CHECK(call)                                      \
    do {                                                      \
        hipError_t err = call;                               \
        if (err != hipSuccess) {                             \
            fprintf(stderr, "CUDA error at %s:%d code=%d (%s)\n", \
                    __FILE__, __LINE__, err, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

// -------------------------------------------------------------
// CUDA Kernel: 使用共享内存的并行求和
// -------------------------------------------------------------
__global__ void reduce_sum_shared(const int* g_input, int* g_output, int N) {
    // 1. 软件层级：Thread/Block 抽象
    // ------------------------------------------------
    // blockDim.x：线程块中的线程数 (Threads per Block)
    // blockIdx.x：当前线程块的索引 (Block Index)
    // threadIdx.x：当前线程在块内的索引 (Thread Index)
    
    // 线程在整个 Grid 中的全局索引
    // 一次kernel调用只有一个grid
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (0 == threadIdx.x && 0 == blockIdx.x) {
        printf("gridx %d, gridy %d gridz %d.\n", gridDim.x, gridDim.y, gridDim.z);
        printf("blocks per grid: %d, threads per block: %d, block index: %d, thread index: %d.\n", gridDim.x, blockDim.x, blockIdx.x, threadIdx.x);
    }
    
    // 线程在块内的局部索引
    int local_tid = threadIdx.x;
    
    // 2. 硬件层级：Shared Memory 和 Bank Conflict 优化
    // ------------------------------------------------
    // __shared__ 声明：数据分配在 SM 上的共享内存中 (非常快速，但容量小)
    // 这种访问模式可以很好地避免 Bank Conflict。
    extern __shared__ int sdata[]; // 动态分配共享内存 (大小在启动时指定)
    //share memory访问延迟约为全局内存（Global Memory）的 1/100，带宽是全局内存的 7~10 倍。

    // 将全局内存数据加载到共享内存
    if (tid < N) {
        sdata[local_tid] = g_input[tid];
    } else {
        // 如果数据量不是 blockDim.x 的整数倍，填充0
        sdata[local_tid] = 0; 
    }
    
    // 同步：确保所有线程都已完成从全局内存到共享内存的加载
    __syncthreads(); 

    // ------------------------------------------------
    // 3. 硬件层级：Warp/SM 的并行归约计算
    // ------------------------------------------------
    // 归约求和：通过并行迭代，将一个 Block 的所有值求和到一个值
    
    // 循环步长 (stride) 从 blockDim.x / 2 开始
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        
        // __syncthreads() 确保每一轮归约完成后，线程才能继续下一轮
        __syncthreads(); 
        
        // 只有线程索引小于步长的线程参与计算
        if (local_tid < stride) {
            // 将 sdata[local_tid + stride] 的值加到 sdata[local_tid]
            // 这就是并行归约的核心操作
            sdata[local_tid] += sdata[local_tid + stride];
            if (0 == threadIdx.x && 0 == blockIdx.x) {
                printf("sum: %d.\n", sdata[0]);
            }
            
            // 硬件配合理解：
            // 第一轮，sdata的后128个各加到前128个。第二轮，sdata的后64个各加到前64个。第三轮后32各加到前32。
            // 在这一步，归约操作由 SM 上的多个 Warp 并行执行。
            // 由于数据在共享内存中，访问速度快，且 sdata[local_tid] 和 sdata[local_tid + stride]
            // 在内存中往往是分散在不同的 Bank，可以实现无 Bank Conflict 的并行访问。
            // 在 NVIDIA GPU 架构中，共享内存被分割成许多独立的物理存储单元，称为 Banks（存储体）。
            // Bank 数量： 现代 NVIDIA GPU 通常将共享内存分割成 32 个或 64 个 Bank。
            // 地址映射： 共享内存中的连续 4 个字节（一个字/word）通常会被映射到 连续的 Bank 中。
            // sdata[local_tid] 和 sdata[local_tid + 1] （即相邻的 4 字节整数）通常属于不同的 Bank。

        }
    }
    // 4. 结果写回：将 Block 结果写回全局内存
    // ------------------------------------------------
    // 只有 Block 的第一个线程 (local_tid == 0) 将最终结果写回全局输出数组
    if (local_tid == 0) {
        // g_output 的大小等于 Grid 的大小（即 Block 的数量）
        g_output[blockIdx.x] = sdata[0];
    }
}

/*
 * NVIDIA Ampere 架构（包括 A100/A800）的共享内存被分割为 32 个 Bank。
 * 这些 Bank 支持一个 Warp（32 个线程）在不发生冲突的情况下，
 * 对连续 32 个 4 字节（32-bit word）的访问，即 32 个线程可以同时访问 32 个不同的 Bank。
 * L1/Shared Memory： Ampere 架构允许每个 Streaming Multiprocessor (SM) 
 * 将 L1 数据缓存和共享内存配置为不同的组合（例如，最高 192KB 的组合容量），但 Bank 的基本数量仍然保持 32 个。
 */
// -------------------------------------------------------------
// Host 代码 (CPU)
// -------------------------------------------------------------
int main() {
    // 数组总大小
    //const int N = 1024 * 1024; 
    const int N = 1024 * 8; 
    // 每个线程块的线程数 (通常是 2 的幂，推荐 256/512/1024)
    const int THREADS_PER_BLOCK = 256; 
    // 计算需要的线程块数
    const int NUM_BLOCKS = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK; 
    
    // 申请主机内存
    int* h_input = (int*)malloc(N * sizeof(int));
    
    // 初始化输入数据
    for (int i = 0; i < N; ++i) {
        h_input[i] = 1; // 方便验证，最终结果应为 N
    }

    // 申请设备内存
    int *d_input, *d_output;
    CUDA_CHECK(hipMalloc((void**)&d_input, N * sizeof(int)));
    CUDA_CHECK(hipMalloc((void**)&d_output, NUM_BLOCKS * sizeof(int)));

    // 将输入数据从主机传输到设备
    CUDA_CHECK(hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice));

    // 启动 Kernel
    // 共享内存大小：每个 Block 需要 THREADS_PER_BLOCK * sizeof(int) 的空间
    size_t shared_mem_size = THREADS_PER_BLOCK * sizeof(int); 
    
    // 启动 Kernel，注意第三个参数是动态分配的共享内存大小
    printf("blocks: %d, threads: %d, sharemem: %lu.\n", NUM_BLOCKS, THREADS_PER_BLOCK, shared_mem_size);
    reduce_sum_shared<<<NUM_BLOCKS, THREADS_PER_BLOCK, shared_mem_size>>>(
        d_input, d_output, N
    );
    CUDA_CHECK(hipGetLastError()); // 检查核函数启动是否成功

    // 等待设备完成计算
    CUDA_CHECK(hipDeviceSynchronize());

    // 将部分结果（每个 Block 的求和结果）从设备传输回主机
    int* h_output = (int*)malloc(NUM_BLOCKS * sizeof(int));
    CUDA_CHECK(hipMemcpy(h_output, d_output, NUM_BLOCKS * sizeof(int), hipMemcpyDeviceToHost));

    // 在主机上对 Block 结果进行最终求和
    long long final_sum = 0;
    for (int i = 0; i < NUM_BLOCKS; ++i) {
        final_sum += h_output[i];
    }

    printf("Input size N = %d\n", N);
    printf("Expected Sum = %d\n", N);
    printf("Calculated Sum = %lld\n", final_sum);

    // 清理资源
    free(h_input);
    free(h_output);
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));

    return 0;
}
