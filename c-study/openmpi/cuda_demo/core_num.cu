#include <stdio.h>
#include <hip/hip_runtime.h>

// 检查 CUDA API 调用是否成功的宏
#define CUDA_CHECK(call)                                      \
    do {                                                      \
        hipError_t err = call;                               \
        if (err != hipSuccess) {                             \
            fprintf(stderr, "CUDA error at %s:%d code=%d (%s)\n", \
                    __FILE__, __LINE__, err, hipGetErrorString(err)); \
            exit(EXIT_FAILURE);                               \
        }                                                     \
    } while (0)

// -------------------------------------------------------------
// 映射函数：根据计算能力（Compute Capability）获取每个SM的CUDA核心数
// -------------------------------------------------------------
// 这是一个必须的映射表，因为不同架构的SM包含的CUDA Core数量不同
int _ConvertSMVer2Cores(int major, int minor) {
    // 资料来源：NVIDIA CUDA C Programming Guide / GPU Architecture
    // 查阅官方文档可以获得最新的和更完整的列表
    
    // (SM version) -> (Cores per SM)
    const int cores_per_sm[] = {
        /* 2.0 (Fermi) */ 32,
        /* 3.0 (Kepler) */ 192,
        /* 5.0 (Maxwell) */ 128,
        /* 6.0 (Pascal) */ 64,
        /* 7.0 (Volta) */ 64,
        /* 7.5 (Turing) */ 64,
        /* 8.0 (Ampere) */ 128,
        /* 8.6 (Ampere) */ 128, // RTX 30系列等
        /* 8.9 (Ada Lovelace) */ 128, // RTX 40系列等
        /* 9.0 (Hopper) */ 128  // H100等
    };

    if (major == 2 && minor == 0) return cores_per_sm[0];
    if (major == 3 && minor == 0) return cores_per_sm[1];
    if (major == 5 && minor == 0) return cores_per_sm[2];
    if (major == 6 && minor == 0) return cores_per_sm[3];
    if (major == 7 && minor == 0) return cores_per_sm[4];
    if (major == 7 && minor == 5) return cores_per_sm[5];
    if (major == 8 && minor == 0) return cores_per_sm[6];
    if (major == 8 && minor == 6) return cores_per_sm[7];
    if (major == 8 && minor == 9) return cores_per_sm[8];
    if (major == 9 && minor == 0) return cores_per_sm[9];

    // Fallback for unknown/future architectures
    fprintf(stderr, "Warning: Unknown SM version %d.%d. Unable to calculate core count.\n", major, minor);
    return 0; 
}


void get_cuda_core_count(int device) {
    hipDeviceProp_t deviceProp;

    // 1. 获取设备属性
    CUDA_CHECK(hipGetDeviceProperties(&deviceProp, device));

    // 2. 获取 SM 数量
    int sm_count = deviceProp.multiProcessorCount;
    
    // 3. 根据计算能力获取每个 SM 的核心数
    int cores_per_sm = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);

    // 4. 计算总的 CUDA Core 数量
    int total_cores = sm_count * cores_per_sm;

    printf("\n--- GPU Device %d: %s ---\n", device, deviceProp.name);
    printf("Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
    printf("Streaming Multiprocessors (SM) Count: %d\n", sm_count);
    printf("CUDA Cores Per SM: %d\n", cores_per_sm);
    printf("Total CUDA Cores: %d\n", total_cores);
}

int main() {
    int device_count;
    
    // 获取设备数量
    CUDA_CHECK(hipGetDeviceCount(&device_count));

    if (device_count == 0) {
        printf("No CUDA devices found.\n");
        return 0;
    }

    printf("Found %d CUDA devices.\n", device_count);

    // 遍历所有 GPU
    for (int i = 0; i < device_count; ++i) {
        get_cuda_core_count(i);
    }

    return 0;
}
